#include "hip/hip_runtime.h"
#include "credit.cuh"

__device__ double RationalApproximation(double t)
{
    double c[] = {2.515517, 0.802853, 0.010328};
    double d[] = {1.432788, 0.189269, 0.001308};
    return t - ((c[2]*t + c[1])*t + c[0]) /
               (((d[2]*t + d[1])*t + d[0])*t + 1.0);
}

__device__ double qNorm(double p)
{
    if ( (p < 0.5) & (p > 0))
    {
        return -RationalApproximation( sqrt(-2.0*log(p)) );
    }
    else if ( (p >= 0.5) & (p < 1))
    {
        return RationalApproximation( sqrt(-2.0*log(1-p)) );
    }
}

Credit::Credit() {}

Credit::~Credit() {}

Credit::Credit(double PD, double EAD, double LGD, double FG, double FL) : PD(PD), EAD(EAD), LGD(LGD), FG(FG), FL(FL)
{
    this->rho = pow(FG, 2) + pow(FL, 2);
    this->FI  = sqrt( 1 - this->rho );
}

__device__ double Credit::loss(double rG, double rL, hiprandState * state)
{
    double Y = sqrt(this->rho) * ( this->FG * rG + FL * rL ) + sqrt( 1 - this->rho ) * hiprand_normal(state);
    return (qNorm(this->PD) >= Y) * EAD * LGD;
}

__device__ Portfolio::Portfolio(Credit *cartera, int n): carte(cartera), n(n) {}

__device__ Portfolio::~Portfolio() {}

__device__ double Portfolio::loss(double fg, double fl, hiprandState *state)
{
    double loss = 0;
    for (int i = 0; i < this->n; i++)
    {
        loss += this->carte[i].loss(fg, fl, state);
    }
    return loss;
}
