#include "hip/hip_runtime.h"
#ifndef __CREDIT_CU__
#define __CREDIT_CU__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "credit.cuh"
#include <iostream>

void handleCudaError(hipError_t cudaERR)
{
    if (cudaERR != hipSuccess)
    {
        printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
    }
}


__global__ void simulation(double * d_out, Credit * cartera, int c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    hiprandState state;
    hiprand_init(12345 + i, 0, 0, &state);

    Portfolio p(cartera, c);

    if (i < n)
    {
        double fg = hiprand_normal(&state);
        double fl = hiprand_normal(&state);

        d_out[i] += p.loss(fg, fl, &state);

        i += gridDim.x * blockDim.x;
    }
}

double * credit_simulation(Credit * cartera, int c, int n)
{
    double * out, * d_out;
    Credit * d_cartera;

    size_t size_d = sizeof(double) * n;
    size_t size_c = sizeof(Credit) * c;

    out = new double[n];
    handleCudaError(hipMalloc(&d_out, size_d));
    handleCudaError(hipMalloc(&d_cartera, size_c));

    handleCudaError(hipMemcpy(d_cartera, cartera, size_c, hipMemcpyHostToDevice));

    simulation<<<0xFFFF, 1024>>>(d_out, d_cartera, c, n);

    handleCudaError(hipMemcpy(out, d_out, size_d, hipMemcpyDeviceToHost));

    handleCudaError(hipFree(d_cartera));
    handleCudaError(hipFree(d_out));

    return out;
}

#endif
